#include "hip/hip_runtime.h"
/*
 *
 *     GNU Radio IEEE 802.11a/g/n/ac 20M bw and upto 2x2
 *     PHY utilization functions and parameters CUDA Version
 *     Copyright (C) Dec 1, 2022  Zelin Yun
 *
 *     This program is free software: you can redistribute it and/or modify
 *     it under the terms of the GNU Affero General Public License as published
 *     by the Free Software Foundation, either version 3 of the License, or
 *     (at your option) any later version.
 *
 *     This program is distributed in the hope that it will be useful,
 *     but WITHOUT ANY WARRANTY; without even the implied warranty of
 *     MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *     GNU Affero General Public License for more details.
 *
 *     You should have received a copy of the GNU Affero General Public License
 *     along with this program.  If not, see <https://www.gnu.org/licenses/>.
 */

#include "cloud80211phycu.cuh"


/*--------------------------------------------------------------------------------------------------------*/

hipFloatComplex* ppSig;
hipFloatComplex* ppSigConj;
hipFloatComplex* ppSigConjAvg;
float* ppSigConjAvgMag;
float* ppSigMag2;
float* ppSigMag2Avg;
float* ppOut;

void preprocMall()
{
  hipMalloc(&ppSig, PREPROC_MAX*sizeof(hipFloatComplex));
  hipMalloc(&ppSigConj, PREPROC_MAX*sizeof(hipFloatComplex));
  hipMalloc(&ppSigConjAvg, PREPROC_MAX*sizeof(hipFloatComplex));
  hipMalloc(&ppSigConjAvgMag, PREPROC_MAX*sizeof(float));
  hipMalloc(&ppSigMag2, PREPROC_MAX*sizeof(float));
  hipMalloc(&ppSigMag2Avg, PREPROC_MAX*sizeof(float));
  hipMalloc(&ppOut, PREPROC_MAX*sizeof(float));
}

void preprocFree()
{
  hipFree(ppSig);
  hipFree(ppSigConj);
  hipFree(ppSigConjAvg);
  hipFree(ppSigConjAvgMag);
  hipFree(ppSigMag2);
  hipFree(ppSigMag2Avg);
  hipFree(ppOut);
}

__global__
void cuPreProcConj(int n, hipFloatComplex* inSig, hipFloatComplex* inSigConj)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i < (n-16))
  {
    inSigConj[i] = hipCmulf(inSig[i], make_hipFloatComplex (hipCrealf(inSig[i+16]), -hipCimagf(inSig[i+16])));
  }
}

__global__
void cuPreProcMag2(int n, hipFloatComplex* inSig, float* inSigMag2)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i < n)
  {
    inSigMag2[i] = hipCabsf(inSig[i]);
    inSigMag2[i] = inSigMag2[i] * inSigMag2[i];
  }
}

__global__
void cuPreProcConjAvgMag(int n, hipFloatComplex* inSigConj, hipFloatComplex* inSigConjAvg, float* inSigConjAvgMag)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i < (n-48))
  {
    inSigConjAvg[i] = make_hipFloatComplex(0.0f, 0.0f);
    for(int j=0;j<48;j++)
    {
      inSigConjAvg[i] = hipCaddf(inSigConjAvg[i], inSigConj[i+j]);
    }
    inSigConjAvgMag[i] = hipCabsf(inSigConjAvg[i]);
  }
}

__global__
void cuPreProcMag2AvgOut(int n, float* inSigMag2, float* inSigMag2Avg, float* inSigConjAvgMag, float* out)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i < (n-64))
  {
    inSigMag2Avg[i] = 0.0f;
    for(int j=0;j<64;j++)
    {
      inSigMag2Avg[i] += inSigMag2[i+j];
    }
    out[i] = inSigConjAvgMag[i] / inSigMag2Avg[i];
  }
}

void cuPreProc(int n, const hipFloatComplex *sig, float* ac, hipFloatComplex* conj)
{
  if(n > 64 && n < PREPROC_MAX)
  {
    hipMemcpy(ppSig, sig, n*sizeof(hipFloatComplex), hipMemcpyHostToDevice);
    cuPreProcConj<<<n/1024 + 1, 1024>>>(n, ppSig, ppSigConj);
    cuPreProcMag2<<<n/1024 + 1, 1024>>>(n, ppSig, ppSigMag2);
    cuPreProcConjAvgMag<<<n/1024 + 1, 1024>>>(n, ppSigConj, ppSigConjAvg, ppSigConjAvgMag);
    cuPreProcMag2AvgOut<<<n/1024 + 1, 1024>>>(n, ppSigMag2, ppSigMag2Avg, ppSigConjAvgMag, ppOut);
    hipMemcpy(ac, ppOut, (n - 64)*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(conj, ppSigConjAvg, (n - 64)*sizeof(hipFloatComplex), hipMemcpyDeviceToHost);
  }
}

/*--------------------------------------------------------------------------------------------------------*/
int mapDeshiftFftLegacy[64] = {
  0, 24, 25, 26, 27, 28, 29, 0, 30, 31, 32, 33, 34, 35, 36, 37, 38, 39, 40, 41, 42, 0, 43, 44, 45, 46, 47, 0, 0, 0, 0, 0, 
  0, 0, 0, 0, 0, 0, 0, 1, 2, 3, 4, 0, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15, 16, 17, 0, 18, 19, 20, 21, 22, 23};
int mapDeshiftFftNonlegacy[64] = {
  0, 26, 27, 28, 29, 30, 31, 0, 32, 33, 34, 35, 36, 37, 38, 39, 40, 41, 42, 43, 44, 0, 45, 46, 47, 48, 49, 50, 51, 0, 0, 0, 
  0, 0, 0, 0, 0, 1, 2, 3, 4, 5, 6, 0, 7, 8, 9, 10, 11, 12, 13, 14, 15, 16, 17, 18, 19, 0, 20, 21, 22, 23, 24, 25};
hipFloatComplex* demodChanSiso;
hipFloatComplex* demodSig;
hipFloatComplex* demodSigFft;
hipfftHandle demodPlan;
float* demodSigLlr;
hipFloatComplex pListTmp[127];
hipFloatComplex* pList;

int* demodDemapFftL;
int* demodDemapBpskL;
int* demodDemapQpskL;
int* demodDemap16QamL;
int* demodDemap64QamL;

int* demodDemapFftNL;
int* demodDemapBpskNL;
int* demodDemapQpskNL;
int* demodDemap16QamNL;
int* demodDemap64QamNL;
int* demodDemap256QamNL;

__global__
void cuDemodChopSamp(int n, hipFloatComplex* sig, hipFloatComplex* sigfft)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = i / 80;       // symbol index
  int offset = i % 80;  
  if(i < n && offset >= 8 && offset < 72)
  {
    sigfft[j*64 + offset - 8] = sig[i];
  }
}

__global__
void cuDemodChanComp(int n, hipFloatComplex* sigfft, hipFloatComplex* chan)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int offset = i % 64;
  if(i < n)
  {
    sigfft[i] = hipCdivf(sigfft[i], chan[offset]);
  }
}

// __global__
// void cuDemodChanComp(int n, hipFloatComplex* sigfft, hipFloatComplex* chan)
// {
//   int i = threadIdx.x;
//   int I = blockIdx.x * blockDim.x + threadIdx.x;
//   int offset = I % 64;
//   __shared__ hipFloatComplex chanIn[64];
//   if(I >= n)
//   {
//     return;
//   }
//   if(i < 64)
//   {
//     chanIn[i] = chan[i];
//   }
//   __syncthreads();
//   sigfft[I] = hipCdivf(sigfft[I], chanIn[offset]);
// }

__global__
void cuDemodQamToLlrLegacy(int n, int nBPSCS, hipFloatComplex* sigfft, float* llr, hipFloatComplex* p, int* deshift, int* deint)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = i / 64;       // sym index
  int offset = i % 64;  // sample index
  int pIndex = (j+1) % 127;
  hipFloatComplex qam;
  float qamReal = 0.0f, qamImag = 0.0f;
  int scIndex = 0;

  __shared__ hipFloatComplex pilot;
  __shared__ float pilotAbs;

  if(i < n)
  {
    return;
  }

  if(offset == 0)
  {
    pilot = make_hipFloatComplex(0.0f, 0.0f);
    pilot = hipCaddf(pilot, hipCmulf(sigfft[j*64 + 43], hipCmulf(make_hipFloatComplex(1.0f, 0.0f), p[pIndex])));
    pilot = hipCaddf(pilot, hipCmulf(sigfft[j*64 + 57], hipCmulf(make_hipFloatComplex(1.0f, 0.0f), p[pIndex])));
    pilot = hipCaddf(pilot, hipCmulf(sigfft[j*64 + 7], hipCmulf(make_hipFloatComplex(1.0f, 0.0f), p[pIndex])));
    pilot = hipCaddf(pilot, hipCmulf(sigfft[j*64 + 21], hipCmulf(make_hipFloatComplex(-1.0f, 0.0f), p[pIndex])));
    pilotAbs = hipCabsf(pilot);
    pilot = hipConjf(pilot);
  }
  __syncthreads();
  if(offset==0 || (offset>=27 && offset<=37) || offset==7 || offset==21 || offset==43 || offset==57)
  {}
  else
  {
    qam = hipCdivf(hipCmulf(sigfft[i], pilot), make_hipFloatComplex(pilotAbs, 0.0f));
    scIndex = deshift[offset];      // sc after fft to data sc index
    if(nBPSCS == 1)
    {
      llr[j*48 + deint[scIndex]] = qamReal;
    }
    else if(nBPSCS == 2)
    {
      qam = hipCmulf(qam, make_hipFloatComplex(1.4142135623730951f, 0.0f));
      qamReal = hipCrealf(qam);
      qamImag = hipCimagf(qam);
      llr[j*96 + deint[scIndex*2]] = qamReal;
      llr[j*96 + deint[scIndex*2+1]] = qamImag;
    }
    else if(nBPSCS == 4)
    {
      qam = hipCmulf(qam, make_hipFloatComplex(3.1622776601683795f, 0.0f));
      qamReal = hipCrealf(qam);
      qamImag = hipCimagf(qam);
      llr[j*192 + deint[scIndex*4]] = qamReal;
      llr[j*192 + deint[scIndex*4+1]] = 2.0f - fabsf(qamReal);
      llr[j*192 + deint[scIndex*4+1]] = qamImag;
      llr[j*192 + deint[scIndex*4+1]] = 2.0f - fabsf(qamImag);
    }
    else
    {
      qam = hipCmulf(qam, make_hipFloatComplex(6.48074069840786f, 0.0f));
      qamReal = hipCrealf(qam);
      qamImag = hipCimagf(qam);
      llr[j*288 + deint[scIndex*6]] = qamReal;
      llr[j*288 + deint[scIndex*6+1]] = 4.0f - fabsf(qamReal);
      llr[j*288 + deint[scIndex*6+2]] = 2.0f - fabsf(4.0f - fabsf(qamReal));
      llr[j*288 + deint[scIndex*6+3]] = qamImag;
      llr[j*288 + deint[scIndex*6+4]] = 4.0f - fabsf(qamImag);
      llr[j*288 + deint[scIndex*6+5]] = 2.0f - fabsf(4.0f - fabsf(qamImag));
    }
  }
}

__global__
void cuDemodQamToLlrNonlegacy(int n, int nBPSCS, hipFloatComplex* sigfft, float* llr, hipFloatComplex* p, int* deshift, int* deint, int format)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = i / 64;       // sym index
  int offset = i % 64;  // sample index
  int pIndex;
  if(format == C8P_F_HT)
  {
    pIndex = (j+3) % 127;
  }
  else
  {
    pIndex = (j+4) % 127;
  }
  hipFloatComplex qam;
  float qamReal = 0.0f, qamImag = 0.0f;
  int scIndex;

  __shared__ hipFloatComplex pilot;
  __shared__ float pilotAbs;

  if(i < n)
  {
    return;
  }

  if(offset == 0)
  {
    float tmpPilot[4];
    if((j%4) == 0)
    {
      tmpPilot[0] = 1.0f; tmpPilot[1] = 1.0f; tmpPilot[2] = 1.0f; tmpPilot[3] = -1.0f;
    }
    else if((j%4) == 1)
    {
      tmpPilot[0] = 1.0f; tmpPilot[1] = 1.0f; tmpPilot[2] = -1.0f; tmpPilot[3] = 1.0f;
    }
    else if((j%4) == 2)
    {
      tmpPilot[0] = 1.0f; tmpPilot[1] = -1.0f; tmpPilot[2] = 1.0f; tmpPilot[3] = 1.0f;
    }
    else
    {
      tmpPilot[0] = -1.0f; tmpPilot[1] = 1.0f; tmpPilot[2] = 1.0f; tmpPilot[3] = 1.0f;
    }
    pilot = make_hipFloatComplex(0.0f, 0.0f);
    pilot = hipCaddf(pilot, hipCmulf(sigfft[j*64 + 43], hipCmulf(make_hipFloatComplex(tmpPilot[0], 0.0f), p[pIndex])));
    pilot = hipCaddf(pilot, hipCmulf(sigfft[j*64 + 57], hipCmulf(make_hipFloatComplex(tmpPilot[1], 0.0f), p[pIndex])));
    pilot = hipCaddf(pilot, hipCmulf(sigfft[j*64 + 7], hipCmulf(make_hipFloatComplex(tmpPilot[2], 0.0f), p[pIndex])));
    pilot = hipCaddf(pilot, hipCmulf(sigfft[j*64 + 21], hipCmulf(make_hipFloatComplex(tmpPilot[3], 0.0f), p[pIndex])));
    pilotAbs = hipCabsf(pilot);
    pilot = hipConjf(pilot);
  }
  __syncthreads();
  if(offset==0 || (offset>=29 && offset<=35) || offset==7 || offset==21 || offset==43 || offset==57)
  {}
  else
  {
    qam = hipCdivf(hipCmulf(sigfft[i], pilot), make_hipFloatComplex(pilotAbs, 0.0f));
    scIndex = deshift[offset];      // sc after fft to data sc index
    if(nBPSCS == 1)
    {
      llr[j*52 + deint[scIndex]] = qamReal;
    }
    else if(nBPSCS == 2)
    {
      qam = hipCmulf(qam, make_hipFloatComplex(1.4142135623730951f, 0.0f));
      qamReal = hipCrealf(qam);
      qamImag = hipCimagf(qam);
      llr[j*104 + deint[scIndex*2]] = qamReal;
      llr[j*104 + deint[scIndex*2+1]] = qamImag;
    }
    else if(nBPSCS == 4)
    {
      qam = hipCmulf(qam, make_hipFloatComplex(3.1622776601683795f, 0.0f));
      qamReal = hipCrealf(qam);
      qamImag = hipCimagf(qam);
      llr[j*208 + deint[scIndex*4]] = qamReal;
      llr[j*208 + deint[scIndex*4+1]] = 2.0f - fabsf(qamReal);
      llr[j*208 + deint[scIndex*4+1]] = qamImag;
      llr[j*208 + deint[scIndex*4+1]] = 2.0f - fabsf(qamImag);
    }
    else if(nBPSCS == 6)
    {
      qam = hipCmulf(qam, make_hipFloatComplex(6.48074069840786f, 0.0f));
      qamReal = hipCrealf(qam);
      qamImag = hipCimagf(qam);
      llr[j*312 + deint[scIndex*6]] = qamReal;
      llr[j*312 + deint[scIndex*6+1]] = 4.0f - fabsf(qamReal);
      llr[j*312 + deint[scIndex*6+2]] = 2.0f - fabsf(4.0f - fabsf(qamReal));
      llr[j*312 + deint[scIndex*6+3]] = qamImag;
      llr[j*312 + deint[scIndex*6+4]] = 4.0f - fabsf(qamImag);
      llr[j*312 + deint[scIndex*6+5]] = 2.0f - fabsf(4.0f - fabsf(qamImag));
    }
    else
    {
      qam = hipCmulf(qam, make_hipFloatComplex(13.038404810405298f, 0.0f));
      qamReal = hipCrealf(qam);
      qamImag = hipCimagf(qam);
      llr[j*416 + deint[scIndex*8]] = qamReal;
      llr[j*416 + deint[scIndex*8+1]] = 8.0f - fabsf(qamReal);
      llr[j*416 + deint[scIndex*8+2]] = 4.0f - fabsf(8.0f - fabsf(qamReal));
      llr[j*416 + deint[scIndex*8+3]] = 2.0f - fabsf(4.0f - fabsf(8.0f - fabsf(qamReal)));
      llr[j*416 + deint[scIndex*8+4]] = qamImag;
      llr[j*416 + deint[scIndex*8+5]] = 8.0f - fabsf(qamImag);
      llr[j*416 + deint[scIndex*8+6]] = 4.0f - fabsf(8.0f - fabsf(qamImag));
      llr[j*416 + deint[scIndex*8+7]] = 2.0f - fabsf(4.0f - fabsf(8.0f - fabsf(qamImag)));
    }
  }
}

void cuDemodMall()
{
  hipMalloc(&demodChanSiso, sizeof(hipFloatComplex) * 64);
  hipMalloc(&demodSig, sizeof(hipFloatComplex) * CUDEMOD_S_MAX * 80);
  hipMalloc(&demodSigFft, sizeof(hipFloatComplex) * CUDEMOD_S_MAX * 64);
  if(hipfftPlan1d(&demodPlan, 64, HIPFFT_C2C, CUDEMOD_FFT_BATCH) != HIPFFT_SUCCESS){
    std::cout<<"cloud80211 cufft, plan creation failed"<<std::endl;
  }
  hipMalloc(&demodSigLlr, sizeof(float) * CUDEMOD_S_MAX * 52 * 8);
  for(int i=0;i<127;i++)
  {
    pListTmp[i] = make_hipFloatComplex((float)PILOT_P[i], 0.0f);
  }
  hipMalloc(&pList, sizeof(hipFloatComplex) * 127);
  hipMemcpy(pList, pListTmp, 127*sizeof(hipFloatComplex), hipMemcpyHostToDevice);
  
  hipMalloc(&demodDemapFftL, sizeof(int) * 64);
  hipMemcpy(demodDemapFftL, mapDeshiftFftLegacy, 64*sizeof(int), hipMemcpyHostToDevice);
  hipMalloc(&demodDemapBpskL, sizeof(int) * 48);
  hipMalloc(&demodDemapQpskL, sizeof(int) * 96);
  hipMalloc(&demodDemap16QamL, sizeof(int) * 192);
  hipMalloc(&demodDemap64QamL, sizeof(int) * 288);

  hipMalloc(&demodDemapFftNL, sizeof(int) * 64);
  hipMemcpy(demodDemapFftNL, mapDeshiftFftNonlegacy, 64*sizeof(int), hipMemcpyHostToDevice);
  hipMalloc(&demodDemapBpskNL, sizeof(int) * 52);
  hipMalloc(&demodDemapQpskNL, sizeof(int) * 104);
  hipMalloc(&demodDemap16QamNL, sizeof(int) * 208);
  hipMalloc(&demodDemap64QamNL, sizeof(int) * 312);
  hipMalloc(&demodDemap256QamNL, sizeof(int) * 416);

  hipMemcpy(demodDemapBpskL, mapDeintLegacyBpsk, 48*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(demodDemapQpskL, mapDeintLegacyQpsk, 96*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(demodDemap16QamL, mapDeintLegacy16Qam, 192*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(demodDemap64QamL, mapDeintLegacy64Qam, 288*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(demodDemapBpskNL, mapDeintNonlegacyBpsk, 52*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(demodDemapQpskNL, mapDeintNonlegacyQpsk, 104*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(demodDemap16QamNL, mapDeintNonlegacy16Qam, 208*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(demodDemap64QamNL, mapDeintNonlegacy64Qam, 312*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(demodDemap256QamNL, mapDeintNonlegacy256Qam, 416*sizeof(int), hipMemcpyHostToDevice);
}

void cuDemodFree()
{
  hipFree(demodChanSiso);
  hipFree(demodSig);
  hipFree(demodSigFft);
  hipfftDestroy(demodPlan);
  hipFree(demodSigLlr);
  hipFree(pList);

  hipFree(demodDemapFftL);
  hipFree(demodDemapBpskL);
  hipFree(demodDemapQpskL);
  hipFree(demodDemap16QamL);
  hipFree(demodDemap64QamL);
  hipFree(demodDemapBpskNL);
  hipFree(demodDemapQpskNL);
  hipFree(demodDemap16QamNL);
  hipFree(demodDemap64QamNL);
  hipFree(demodDemap256QamNL);
}

void cuDemodChanSiso(hipFloatComplex *chan)
{
  hipMemcpy(demodChanSiso, chan, 64*sizeof(hipFloatComplex), hipMemcpyHostToDevice);
}

void cuDemodSigCopy(int i, int n, const hipFloatComplex *sig)
{
  if(i >= 0 && n >= 0 && (i+n) < (CUDEMOD_S_MAX * 80))
  {
    hipMemcpy(&demodSig[i], sig, n*sizeof(hipFloatComplex), hipMemcpyHostToDevice);
  }
}

void cuDemodSiso(c8p_mod* m)
{
  cuDemodChopSamp<<<(m->nSym * m->nSymSamp)/1024 + 1, 1024>>>(m->nSym * m->nSymSamp, demodSig, demodSigFft);
  for(int symIter=0; symIter < ((m->nSym + CUDEMOD_FFT_BATCH - 1) / CUDEMOD_FFT_BATCH); symIter++ )   // each round inlcudes 256 batches
  {
    hipfftExecC2C(demodPlan, &demodSigFft[symIter*CUDEMOD_FFT_BATCH*64], &demodSigFft[symIter*CUDEMOD_FFT_BATCH*64], HIPFFT_FORWARD);
  }
  cuDemodChanComp<<<(m->nSym * 64)/1024 + 1, 1024>>>(m->nSym * 64, demodSigFft, demodChanSiso);
  if(m->format == C8P_F_L)
  {
    if(m->nBPSCS == 1)
    {
      cuDemodQamToLlrLegacy<<<(m->nSym * 64)/1024 + 1, 1024>>>(m->nSym * 64, m->nBPSCS, demodSigFft, demodSigLlr, pList, demodDemapFftL, demodDemapBpskL);
    }
    else if(m->nBPSCS == 2)
    {
      cuDemodQamToLlrLegacy<<<(m->nSym * 64)/1024 + 1, 1024>>>(m->nSym * 64, m->nBPSCS, demodSigFft, demodSigLlr, pList, demodDemapFftL, demodDemapQpskL);
    }
    else if(m->nBPSCS == 4)
    {
      cuDemodQamToLlrLegacy<<<(m->nSym * 64)/1024 + 1, 1024>>>(m->nSym * 64, m->nBPSCS, demodSigFft, demodSigLlr, pList, demodDemapFftL, demodDemap16QamL);
    }
    else
    {
      cuDemodQamToLlrLegacy<<<(m->nSym * 64)/1024 + 1, 1024>>>(m->nSym * 64, m->nBPSCS, demodSigFft, demodSigLlr, pList, demodDemapFftL, demodDemap64QamL);
    }
  }
  else
  {
    if(m->nBPSCS == 1)
    {
      cuDemodQamToLlrNonlegacy<<<(m->nSym * 64)/1024 + 1, 1024>>>(m->nSym * 64, m->nBPSCS, demodSigFft, demodSigLlr, pList, demodDemapFftL, demodDemapBpskNL, m->format);
    }
    else if(m->nBPSCS == 2)
    {
      cuDemodQamToLlrNonlegacy<<<(m->nSym * 64)/1024 + 1, 1024>>>(m->nSym * 64, m->nBPSCS, demodSigFft, demodSigLlr, pList, demodDemapFftL, demodDemapQpskNL, m->format);
    }
    else if(m->nBPSCS == 4)
    {
      cuDemodQamToLlrNonlegacy<<<(m->nSym * 64)/1024 + 1, 1024>>>(m->nSym * 64, m->nBPSCS, demodSigFft, demodSigLlr, pList, demodDemapFftL, demodDemap16QamNL, m->format);
    }
    else if(m->nBPSCS == 6)
    {
      cuDemodQamToLlrNonlegacy<<<(m->nSym * 64)/1024 + 1, 1024>>>(m->nSym * 64, m->nBPSCS, demodSigFft, demodSigLlr, pList, demodDemapFftL, demodDemap64QamNL, m->format);
    }
    else
    {
      cuDemodQamToLlrNonlegacy<<<(m->nSym * 64)/1024 + 1, 1024>>>(m->nSym * 64, m->nBPSCS, demodSigFft, demodSigLlr, pList, demodDemapFftL, demodDemap64QamNL, m->format);
    }
  }
}

void cuDemodDebug(int n, hipFloatComplex* outcomp, int m, float* outfloat)
{
  hipMemcpy(outcomp, demodSigFft, n*sizeof(hipFloatComplex), hipMemcpyDeviceToHost);
  // hipMemcpy(outfloat, demodSigFft, m*sizeof(float), hipMemcpyDeviceToHost);
}