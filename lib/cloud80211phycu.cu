#include "hip/hip_runtime.h"
/*
 *
 *     GNU Radio IEEE 802.11a/g/n/ac 20M bw and upto 2x2
 *     PHY utilization functions and parameters CUDA Version
 *     Copyright (C) Dec 1, 2022  Zelin Yun
 *
 *     This program is free software: you can redistribute it and/or modify
 *     it under the terms of the GNU Affero General Public License as published
 *     by the Free Software Foundation, either version 3 of the License, or
 *     (at your option) any later version.
 *
 *     This program is distributed in the hope that it will be useful,
 *     but WITHOUT ANY WARRANTY; without even the implied warranty of
 *     MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *     GNU Affero General Public License for more details.
 *
 *     You should have received a copy of the GNU Affero General Public License
 *     along with this program.  If not, see <https://www.gnu.org/licenses/>.
 */

#include "cloud80211phycu.cuh"


/*--------------------------------------------------------------------------------------------------------*/

hipFloatComplex* ppSig;
hipFloatComplex* ppSigConj;
hipFloatComplex* ppSigConjAvg;
float* ppSigConjAvgMag;
float* ppSigMag2;
float* ppSigMag2Avg;
float* ppOut;

void preprocMall()
{
  hipMalloc(&ppSig, PREPROC_MAX*sizeof(hipFloatComplex));
  hipMalloc(&ppSigConj, PREPROC_MAX*sizeof(hipFloatComplex));
  hipMalloc(&ppSigConjAvg, PREPROC_MAX*sizeof(hipFloatComplex));
  hipMalloc(&ppSigConjAvgMag, PREPROC_MAX*sizeof(float));
  hipMalloc(&ppSigMag2, PREPROC_MAX*sizeof(float));
  hipMalloc(&ppSigMag2Avg, PREPROC_MAX*sizeof(float));
  hipMalloc(&ppOut, PREPROC_MAX*sizeof(float));
}

void preprocFree()
{
  hipFree(ppSig);
  hipFree(ppSigConj);
  hipFree(ppSigConjAvg);
  hipFree(ppSigConjAvgMag);
  hipFree(ppSigMag2);
  hipFree(ppSigMag2Avg);
  hipFree(ppOut);
}

__global__
void cuPreProcConj(int n, hipFloatComplex* inSig, hipFloatComplex* inSigConj)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i < (n-16))
  {
    inSigConj[i] = hipCmulf(inSig[i], make_hipFloatComplex (hipCrealf(inSig[i+16]), -hipCimagf(inSig[i+16])));
  }
}

__global__
void cuPreProcMag2(int n, hipFloatComplex* inSig, float* inSigMag2)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i < n)
  {
    inSigMag2[i] = hipCabsf(inSig[i]);
    inSigMag2[i] = inSigMag2[i] * inSigMag2[i];
  }
}

__global__
void cuPreProcConjAvgMag(int n, hipFloatComplex* inSigConj, hipFloatComplex* inSigConjAvg, float* inSigConjAvgMag)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i < (n-48))
  {
    inSigConjAvg[i] = make_hipFloatComplex(0.0f, 0.0f);
    for(int j=0;j<48;j++)
    {
      inSigConjAvg[i] = hipCaddf(inSigConjAvg[i], inSigConj[i+j]);
    }
    inSigConjAvgMag[i] = hipCabsf(inSigConjAvg[i]);
  }
}

__global__
void cuPreProcMag2AvgOut(int n, float* inSigMag2, float* inSigMag2Avg, float* inSigConjAvgMag, float* out)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i < (n-64))
  {
    inSigMag2Avg[i] = 0.0f;
    for(int j=0;j<64;j++)
    {
      inSigMag2Avg[i] += inSigMag2[i+j];
    }
    out[i] = inSigConjAvgMag[i] / inSigMag2Avg[i];
  }
}

void cuPreProc(int n, const hipFloatComplex *sig, float* ac, hipFloatComplex* conj)
{
  if(n > 64 && n < PREPROC_MAX)
  {
    hipMemcpy(ppSig, sig, n*sizeof(hipFloatComplex), hipMemcpyHostToDevice);
    cuPreProcConj<<<n/1024 + 1, 1024>>>(n, ppSig, ppSigConj);
    cuPreProcMag2<<<n/1024 + 1, 1024>>>(n, ppSig, ppSigMag2);
    cuPreProcConjAvgMag<<<n/1024 + 1, 1024>>>(n, ppSigConj, ppSigConjAvg, ppSigConjAvgMag);
    cuPreProcMag2AvgOut<<<n/1024 + 1, 1024>>>(n, ppSigMag2, ppSigMag2Avg, ppSigConjAvgMag, ppOut);
    hipMemcpy(ac, ppOut, (n - 64)*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(conj, ppSigConjAvg, (n - 64)*sizeof(hipFloatComplex), hipMemcpyDeviceToHost);
  }
}

/*--------------------------------------------------------------------------------------------------------*/
int mapDeshiftFftLegacy[64] = {
  -1, 24, 25, 26, 27, 28, 29, -1, 30, 31, 32, 33, 34, 35, 36, 37, 38, 39, 40, 41, 42, -1, 43, 44, 45, 46, 47, -1, -1, -1, -1, -1, 
  -1, -1, -1, -1, -1, -1, 0, 1, 2, 3, 4, -1, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15, 16, 17, -1, 18, 19, 20, 21, 22, 23};
int mapDeshiftFftNonlegacy[64] = {
  -1, 26, 27, 28, 29, 30, 31, -1, 32, 33, 34, 35, 36, 37, 38, 39, 40, 41, 42, 43, 44, -1, 45, 46, 47, 48, 49, 50, 51, -1, -1, -1, 
  -1, -1, -1, -1, 0, 1, 2, 3, 4, 5, 6, -1, 7, 8, 9, 10, 11, 12, 13, 14, 15, 16, 17, 18, 19, -1, 20, 21, 22, 23, 24, 25};
hipFloatComplex* demodChanSiso;
hipFloatComplex* demodSig;
hipFloatComplex* demodSigFft;
hipfftHandle demodPlan;
float* demodSigLlr;
hipFloatComplex* pilotsLegacy;
hipFloatComplex* pilotsHt;
hipFloatComplex* pilotsVht;

int* demodDemapFftL;
int* demodDemapBpskL;
int* demodDemapQpskL;
int* demodDemap16QamL;
int* demodDemap64QamL;

int* demodDemapFftNL;
int* demodDemapBpskNL;
int* demodDemapQpskNL;
int* demodDemap16QamNL;
int* demodDemap64QamNL;
int* demodDemap256QamNL;

__global__
void cuDemodChopSamp(int n, hipFloatComplex* sig, hipFloatComplex* sigfft)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = i / 80;       // symbol index
  int offset = i % 80;  
  if(i < n && offset >= 8 && offset < 72)
  {
    sigfft[j*64 + offset - 8] = sig[i];
  }
}

__global__
void cuDemodChanComp(int n, hipFloatComplex* sigfft, hipFloatComplex* chan)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int offset = i % 64;
  if(i < n)
  {
    sigfft[i] = hipCdivf(sigfft[i], chan[offset]);
  }
}

// __global__
// void cuDemodChanComp(int n, hipFloatComplex* sigfft, hipFloatComplex* chan)
// {
//   int i = threadIdx.x;
//   int I = blockIdx.x * blockDim.x + threadIdx.x;
//   int offset = I % 64;
//   __shared__ hipFloatComplex chanIn[64];
//   if(I >= n)
//   {
//     return;
//   }
//   if(i < 64)
//   {
//     chanIn[i] = chan[i];
//   }
//   __syncthreads();
//   sigfft[I] = hipCdivf(sigfft[I], chanIn[offset]);
// }

__global__
void cuDemodQamToLlr(int n, int nCBPSS, hipFloatComplex* sigfft, float* llr, hipFloatComplex* p, int* deshift, int* deint)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = i / 64;  // sym index
  int k = i % 64;  // sample index
  int llrOffset = j * nCBPSS;
  hipFloatComplex pilotConj = make_hipFloatComplex(0.0f, 0.0f);
  hipFloatComplex pilotAbs;
  hipFloatComplex qam;
  float qamReal, qamImag;
  int scIndex = deshift[k];      // sc after fft to data sc index

  if(i >= n || scIndex < 0)
  {
    return;
  }

  pilotConj = hipCaddf(pilotConj, hipCmulf(sigfft[j*64 + 43], p[j*4]));
  pilotConj = hipCaddf(pilotConj, hipCmulf(sigfft[j*64 + 57], p[j*4+1]));
  pilotConj = hipCaddf(pilotConj, hipCmulf(sigfft[j*64 +  7], p[j*4+2]));
  pilotConj = hipCaddf(pilotConj, hipCmulf(sigfft[j*64 + 21], p[j*4+3]));
  pilotAbs = make_hipFloatComplex(hipCabsf(pilotConj), 0.0f);
  pilotConj = hipConjf(pilotConj);

  qam = hipCdivf(hipCmulf(sigfft[i], pilotConj), pilotAbs);

  if(nCBPSS == 48 || nCBPSS == 52)
  {
    qamReal = hipCrealf(qam);
    qamImag = hipCimagf(qam);
    llr[llrOffset + deint[scIndex]] = qamReal;
  }
  else if(nCBPSS == 96 || nCBPSS == 104)
  {
    qam = hipCmulf(qam, make_hipFloatComplex(1.4142135623730951f, 0.0f));
    qamReal = hipCrealf(qam);
    qamImag = hipCimagf(qam);
    llr[llrOffset + deint[scIndex*2]] = qamReal;
    llr[llrOffset + deint[scIndex*2+1]] = qamImag;
  }
  else if(nCBPSS == 192 || nCBPSS == 208)
  {
    qam = hipCmulf(qam, make_hipFloatComplex(3.1622776601683795f, 0.0f));
    qamReal = hipCrealf(qam);
    qamImag = hipCimagf(qam);
    llr[llrOffset + deint[scIndex*4]] = qamReal;
    llr[llrOffset + deint[scIndex*4+1]] = 2.0f - fabsf(qamReal);
    llr[llrOffset + deint[scIndex*4+2]] = qamImag;
    llr[llrOffset + deint[scIndex*4+3]] = 2.0f - fabsf(qamImag);
  }
  else if(nCBPSS == 288 || nCBPSS == 312)
  {
    qam = hipCmulf(qam, make_hipFloatComplex(6.48074069840786f, 0.0f));
    qamReal = hipCrealf(qam);
    qamImag = hipCimagf(qam);
    llr[llrOffset + deint[scIndex*6]] = qamReal;
    llr[llrOffset + deint[scIndex*6+1]] = 4.0f - fabsf(qamReal);
    llr[llrOffset + deint[scIndex*6+2]] = 2.0f - fabsf(4.0f - fabsf(qamReal));
    llr[llrOffset + deint[scIndex*6+3]] = qamImag;
    llr[llrOffset + deint[scIndex*6+4]] = 4.0f - fabsf(qamImag);
    llr[llrOffset + deint[scIndex*6+5]] = 2.0f - fabsf(4.0f - fabsf(qamImag));
  }
  else
  {
    qam = hipCmulf(qam, make_hipFloatComplex(13.038404810405298f, 0.0f));
    qamReal = hipCrealf(qam);
    qamImag = hipCimagf(qam);
    llr[llrOffset + deint[scIndex*8]] = qamReal;
    llr[llrOffset + deint[scIndex*8+1]] = 8.0f - fabsf(qamReal);
    llr[llrOffset + deint[scIndex*8+2]] = 4.0f - fabsf(8.0f - fabsf(qamReal));
    llr[llrOffset + deint[scIndex*8+3]] = 2.0f - fabsf(4.0f - fabsf(8.0f - fabsf(qamReal)));
    llr[llrOffset + deint[scIndex*8+4]] = qamImag;
    llr[llrOffset + deint[scIndex*8+5]] = 8.0f - fabsf(qamImag);
    llr[llrOffset + deint[scIndex*8+6]] = 4.0f - fabsf(8.0f - fabsf(qamImag));
    llr[llrOffset + deint[scIndex*8+7]] = 2.0f - fabsf(4.0f - fabsf(8.0f - fabsf(qamImag)));
  }
  // sigfft[i] = make_hipFloatComplex(llrOffset + scIndex*2, llrOffset + scIndex*2 + 1);
}

void cuDemodMall()
{
  hipMalloc(&demodChanSiso, sizeof(hipFloatComplex) * 64);
  hipMalloc(&demodSig, sizeof(hipFloatComplex) * CUDEMOD_S_MAX * 80);
  hipMalloc(&demodSigFft, sizeof(hipFloatComplex) * CUDEMOD_S_MAX * 64);
  if(hipfftPlan1d(&demodPlan, 64, HIPFFT_C2C, CUDEMOD_FFT_BATCH) != HIPFFT_SUCCESS){
    std::cout<<"cloud80211 cufft, plan creation failed"<<std::endl;
  }
  hipMalloc(&demodSigLlr, sizeof(float) * CUDEMOD_S_MAX * 52 * 8);
  hipMemset(demodSigLlr, 0, sizeof(float) * CUDEMOD_S_MAX * 52 * 8);

  hipFloatComplex pListTmp[CUDEMOD_S_MAX * 4];
  hipMalloc(&pilotsLegacy, sizeof(hipFloatComplex) * CUDEMOD_S_MAX * 4);
  hipMalloc(&pilotsHt, sizeof(hipFloatComplex) * CUDEMOD_S_MAX * 4);
  hipMalloc(&pilotsVht, sizeof(hipFloatComplex) * CUDEMOD_S_MAX * 4);
  for(int i=0;i<CUDEMOD_S_MAX;i++)
  {
    pListTmp[i*4] = make_hipFloatComplex(1.0f * PILOT_P[(i+1)%127], 0.0f);
    pListTmp[i*4+1] = make_hipFloatComplex(1.0f * PILOT_P[(i+1)%127], 0.0f);
    pListTmp[i*4+2] = make_hipFloatComplex(1.0f * PILOT_P[(i+1)%127], 0.0f);
    pListTmp[i*4+3] = make_hipFloatComplex(-1.0f * PILOT_P[(i+1)%127], 0.0f);
  }
  hipMemcpy(pilotsLegacy, pListTmp, sizeof(hipFloatComplex) * CUDEMOD_S_MAX * 4, hipMemcpyHostToDevice);
  float pTmp[4] = {1.0f, 1.0f, 1.0f, -1.0f};
  for(int i=0;i<CUDEMOD_S_MAX;i++)
  {
    pListTmp[i*4] = make_hipFloatComplex(pTmp[0] * PILOT_P[(i+3)%127], 0.0f);
    pListTmp[i*4+1] = make_hipFloatComplex(pTmp[1] * PILOT_P[(i+3)%127], 0.0f);
    pListTmp[i*4+2] = make_hipFloatComplex(pTmp[2] * PILOT_P[(i+3)%127], 0.0f);
    pListTmp[i*4+3] = make_hipFloatComplex(pTmp[3] * PILOT_P[(i+3)%127], 0.0f);

    float tmpPilot = pTmp[0];
    pTmp[0] = pTmp[1];
    pTmp[1] = pTmp[2];
    pTmp[2] = pTmp[3];
    pTmp[3] = tmpPilot;
  }
  hipMemcpy(pilotsHt, pListTmp, sizeof(hipFloatComplex) * CUDEMOD_S_MAX * 4, hipMemcpyHostToDevice);

  float pTmp2[4] = {1.0f, 1.0f, 1.0f, -1.0f};
  for(int i=0;i<CUDEMOD_S_MAX;i++)
  {
    pListTmp[i*4] = make_hipFloatComplex(pTmp2[0] * PILOT_P[(i+4)%127], 0.0f);
    pListTmp[i*4+1] = make_hipFloatComplex(pTmp2[1] * PILOT_P[(i+4)%127], 0.0f);
    pListTmp[i*4+2] = make_hipFloatComplex(pTmp2[2] * PILOT_P[(i+4)%127], 0.0f);
    pListTmp[i*4+3] = make_hipFloatComplex(pTmp2[3] * PILOT_P[(i+4)%127], 0.0f);

    float tmpPilot = pTmp2[0];
    pTmp2[0] = pTmp2[1];
    pTmp2[1] = pTmp2[2];
    pTmp2[2] = pTmp2[3];
    pTmp2[3] = tmpPilot;
  }
  hipMemcpy(pilotsVht, pListTmp, sizeof(hipFloatComplex) * CUDEMOD_S_MAX * 4, hipMemcpyHostToDevice);
  
  
  hipMalloc(&demodDemapFftL, sizeof(int) * 64);
  hipMemcpy(demodDemapFftL, mapDeshiftFftLegacy, 64*sizeof(int), hipMemcpyHostToDevice);
  hipMalloc(&demodDemapBpskL, sizeof(int) * 48);
  hipMalloc(&demodDemapQpskL, sizeof(int) * 96);
  hipMalloc(&demodDemap16QamL, sizeof(int) * 192);
  hipMalloc(&demodDemap64QamL, sizeof(int) * 288);

  hipMalloc(&demodDemapFftNL, sizeof(int) * 64);
  hipMemcpy(demodDemapFftNL, mapDeshiftFftNonlegacy, 64*sizeof(int), hipMemcpyHostToDevice);
  hipMalloc(&demodDemapBpskNL, sizeof(int) * 52);
  hipMalloc(&demodDemapQpskNL, sizeof(int) * 104);
  hipMalloc(&demodDemap16QamNL, sizeof(int) * 208);
  hipMalloc(&demodDemap64QamNL, sizeof(int) * 312);
  hipMalloc(&demodDemap256QamNL, sizeof(int) * 416);

  hipMemcpy(demodDemapBpskL, mapDeintLegacyBpsk, 48*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(demodDemapQpskL, mapDeintLegacyQpsk, 96*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(demodDemap16QamL, mapDeintLegacy16Qam, 192*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(demodDemap64QamL, mapDeintLegacy64Qam, 288*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(demodDemapBpskNL, mapDeintNonlegacyBpsk, 52*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(demodDemapQpskNL, mapDeintNonlegacyQpsk, 104*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(demodDemap16QamNL, mapDeintNonlegacy16Qam, 208*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(demodDemap64QamNL, mapDeintNonlegacy64Qam, 312*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(demodDemap256QamNL, mapDeintNonlegacy256Qam, 416*sizeof(int), hipMemcpyHostToDevice);
}

void cuDemodFree()
{
  hipFree(demodChanSiso);
  hipFree(demodSig);
  hipFree(demodSigFft);
  hipfftDestroy(demodPlan);
  hipFree(demodSigLlr);
  hipFree(pilotsLegacy);
  hipFree(pilotsHt);
  hipFree(pilotsVht);

  hipFree(demodDemapFftL);
  hipFree(demodDemapBpskL);
  hipFree(demodDemapQpskL);
  hipFree(demodDemap16QamL);
  hipFree(demodDemap64QamL);
  hipFree(demodDemapBpskNL);
  hipFree(demodDemapQpskNL);
  hipFree(demodDemap16QamNL);
  hipFree(demodDemap64QamNL);
  hipFree(demodDemap256QamNL);
}

void cuDemodChanSiso(hipFloatComplex *chan)
{
  hipMemcpy(demodChanSiso, chan, 64*sizeof(hipFloatComplex), hipMemcpyHostToDevice);
}

void cuDemodSigCopy(int i, int n, const hipFloatComplex *sig)
{
  if(i >= 0 && n >= 0 && (i+n) < (CUDEMOD_S_MAX * 80))
  {
    hipMemcpy(&demodSig[i], sig, n*sizeof(hipFloatComplex), hipMemcpyHostToDevice);
  }
}

void cuDemodSiso(c8p_mod* m)
{
  cuDemodChopSamp<<<(m->nSym * m->nSymSamp)/1024 + 1, 1024>>>(m->nSym * m->nSymSamp, demodSig, demodSigFft);
  for(int symIter=0; symIter < ((m->nSym + CUDEMOD_FFT_BATCH - 1) / CUDEMOD_FFT_BATCH); symIter++ )   // each round inlcudes 256 batches
  {
    hipfftExecC2C(demodPlan, &demodSigFft[symIter*CUDEMOD_FFT_BATCH*64], &demodSigFft[symIter*CUDEMOD_FFT_BATCH*64], HIPFFT_FORWARD);
  }
  cuDemodChanComp<<<(m->nSym * 64)/1024 + 1, 1024>>>(m->nSym * 64, demodSigFft, demodChanSiso);
  if(m->format == C8P_F_L)
  {
    if(m->mod == C8P_QAM_BPSK)
    {
      cuDemodQamToLlr<<<(m->nSym * 64)/256 + 1, 256>>>(m->nSym * 64, m->nCBPSS, demodSigFft, demodSigLlr, pilotsLegacy, demodDemapFftL, demodDemapBpskL);
    }
    else if(m->mod == C8P_QAM_QPSK)
    {
      cuDemodQamToLlr<<<(m->nSym * 64)/256 + 1, 256>>>(m->nSym * 64, m->nCBPSS, demodSigFft, demodSigLlr, pilotsLegacy, demodDemapFftL, demodDemapQpskL);
    }
    else if(m->mod == C8P_QAM_16QAM)
    {
      cuDemodQamToLlr<<<(m->nSym * 64)/256 + 1, 256>>>(m->nSym * 64, m->nCBPSS, demodSigFft, demodSigLlr, pilotsLegacy, demodDemapFftL, demodDemap16QamL);
    }
    else
    {
      cuDemodQamToLlr<<<(m->nSym * 64)/256 + 1, 256>>>(m->nSym * 64, m->nCBPSS, demodSigFft, demodSigLlr, pilotsLegacy, demodDemapFftL, demodDemap64QamL);
    }
  }
  else
  {
    if(m->mod == C8P_QAM_BPSK)
    {
      if(m->format == C8P_F_HT)
      {cuDemodQamToLlr<<<(m->nSym * 64)/1024 + 1, 1024>>>(m->nSym * 64, m->nCBPSS, demodSigFft, demodSigLlr, pilotsHt, demodDemapFftNL, demodDemapBpskNL);}
      else
      {cuDemodQamToLlr<<<(m->nSym * 64)/1024 + 1, 1024>>>(m->nSym * 64, m->nCBPSS, demodSigFft, demodSigLlr, pilotsVht, demodDemapFftNL, demodDemapBpskNL);}
    }
    else if(m->mod == C8P_QAM_QPSK)
    {
      if(m->format == C8P_F_HT)
      {cuDemodQamToLlr<<<(m->nSym * 64)/1024 + 1, 1024>>>(m->nSym * 64, m->nCBPSS, demodSigFft, demodSigLlr, pilotsHt, demodDemapFftNL, demodDemapQpskNL);}
      else
      {cuDemodQamToLlr<<<(m->nSym * 64)/1024 + 1, 1024>>>(m->nSym * 64, m->nCBPSS, demodSigFft, demodSigLlr, pilotsVht, demodDemapFftNL, demodDemapQpskNL);}
    }
    else if(m->mod == C8P_QAM_16QAM)
    {
      if(m->format == C8P_F_HT)
      {cuDemodQamToLlr<<<(m->nSym * 64)/1024 + 1, 1024>>>(m->nSym * 64, m->nCBPSS, demodSigFft, demodSigLlr, pilotsHt, demodDemapFftNL, demodDemap16QamNL);}
      else
      {cuDemodQamToLlr<<<(m->nSym * 64)/1024 + 1, 1024>>>(m->nSym * 64, m->nCBPSS, demodSigFft, demodSigLlr, pilotsVht, demodDemapFftNL, demodDemap16QamNL);}
    }
    else if(m->mod == C8P_QAM_64QAM)
    {
      if(m->format == C8P_F_HT)
      {cuDemodQamToLlr<<<(m->nSym * 64)/1024 + 1, 1024>>>(m->nSym * 64, m->nCBPSS, demodSigFft, demodSigLlr, pilotsHt, demodDemapFftNL, demodDemap64QamNL);}
      else
      {cuDemodQamToLlr<<<(m->nSym * 64)/1024 + 1, 1024>>>(m->nSym * 64, m->nCBPSS, demodSigFft, demodSigLlr, pilotsVht, demodDemapFftNL, demodDemap64QamNL);}
    }
    else
    {
      cuDemodQamToLlr<<<(m->nSym * 64)/1024 + 1, 1024>>>(m->nSym * 64, m->nCBPSS, demodSigFft, demodSigLlr, pilotsVht, demodDemapFftNL, demodDemap256QamNL);
    }
  }
}

void cuDemodDebug(int n, hipFloatComplex* outcomp, int m, float* outfloat)
{
  hipMemcpy(outcomp, demodSigFft, n*sizeof(hipFloatComplex), hipMemcpyDeviceToHost);
  hipMemcpy(outfloat, demodSigLlr, m*sizeof(float), hipMemcpyDeviceToHost);
}