#include "hip/hip_runtime.h"
/*
 *
 *     GNU Radio IEEE 802.11a/g/n/ac 20M bw and upto 2x2
 *     PHY utilization functions and parameters CUDA Version
 *     Copyright (C) Dec 1, 2022  Zelin Yun
 *
 *     This program is free software: you can redistribute it and/or modify
 *     it under the terms of the GNU Affero General Public License as published
 *     by the Free Software Foundation, either version 3 of the License, or
 *     (at your option) any later version.
 *
 *     This program is distributed in the hope that it will be useful,
 *     but WITHOUT ANY WARRANTY; without even the implied warranty of
 *     MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *     GNU Affero General Public License for more details.
 *
 *     You should have received a copy of the GNU Affero General Public License
 *     along with this program.  If not, see <https://www.gnu.org/licenses/>.
 */

#include "cloud80211phycu.cuh"


/*--------------------------------------------------------------------------------------------------------*/

hipFloatComplex* ppSig;
hipFloatComplex* ppSigConj;
hipFloatComplex* ppSigConjAvg;
float* ppSigConjAvgMag;
float* ppSigMag2;
float* ppSigMag2Avg;
float* ppOut;

void preprocMall()
{
  hipMalloc(&ppSig, PREPROC_MAX*sizeof(hipFloatComplex));
  hipMalloc(&ppSigConj, PREPROC_MAX*sizeof(hipFloatComplex));
  hipMalloc(&ppSigConjAvg, PREPROC_MAX*sizeof(hipFloatComplex));
  hipMalloc(&ppSigConjAvgMag, PREPROC_MAX*sizeof(float));
  hipMalloc(&ppSigMag2, PREPROC_MAX*sizeof(float));
  hipMalloc(&ppSigMag2Avg, PREPROC_MAX*sizeof(float));
  hipMalloc(&ppOut, PREPROC_MAX*sizeof(float));
}

void preprocFree()
{
  hipFree(ppSig);
  hipFree(ppSigConj);
  hipFree(ppSigConjAvg);
  hipFree(ppSigConjAvgMag);
  hipFree(ppSigMag2);
  hipFree(ppSigMag2Avg);
  hipFree(ppOut);
}

__global__
void cuPreProcConj(int n, hipFloatComplex* inSig, hipFloatComplex* inSigConj)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i < (n-16))
  {
    inSigConj[i] = hipCmulf(inSig[i], make_hipFloatComplex (hipCrealf(inSig[i+16]), -hipCimagf(inSig[i+16])));
  }
}

__global__
void cuPreProcMag2(int n, hipFloatComplex* inSig, float* inSigMag2)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i < n)
  {
    inSigMag2[i] = hipCabsf(inSig[i]);
    inSigMag2[i] = inSigMag2[i] * inSigMag2[i];
  }
}

__global__
void cuPreProcConjAvgMag(int n, hipFloatComplex* inSigConj, hipFloatComplex* inSigConjAvg, float* inSigConjAvgMag)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i < (n-48))
  {
    inSigConjAvg[i] = make_hipFloatComplex(0.0f, 0.0f);
    for(int j=0;j<48;j++)
    {
      inSigConjAvg[i] = hipCaddf(inSigConjAvg[i], inSigConj[i+j]);
    }
    inSigConjAvgMag[i] = hipCabsf(inSigConjAvg[i]);
  }
}

__global__
void cuPreProcMag2AvgOut(int n, float* inSigMag2, float* inSigMag2Avg, float* inSigConjAvgMag, float* out)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i < (n-64))
  {
    inSigMag2Avg[i] = 0.0f;
    for(int j=0;j<64;j++)
    {
      inSigMag2Avg[i] += inSigMag2[i+j];
    }
    out[i] = inSigConjAvgMag[i] / inSigMag2Avg[i];
  }
}

void cuPreProc(int n, const hipFloatComplex *sig, float* ac, hipFloatComplex* conj)
{
  if(n > 64 && n < PREPROC_MAX)
  {
    hipMemcpy(ppSig, sig, n*sizeof(hipFloatComplex), hipMemcpyHostToDevice);
    cuPreProcConj<<<n/1024 + 1, 1024>>>(n, ppSig, ppSigConj);
    cuPreProcMag2<<<n/1024 + 1, 1024>>>(n, ppSig, ppSigMag2);
    cuPreProcConjAvgMag<<<n/1024 + 1, 1024>>>(n, ppSigConj, ppSigConjAvg, ppSigConjAvgMag);
    cuPreProcMag2AvgOut<<<n/1024 + 1, 1024>>>(n, ppSigMag2, ppSigMag2Avg, ppSigConjAvgMag, ppOut);
    hipMemcpy(ac, ppOut, (n - 64)*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(conj, ppSigConjAvg, (n - 64)*sizeof(hipFloatComplex), hipMemcpyDeviceToHost);
  }
}

/*--------------------------------------------------------------------------------------------------------*/
hipFloatComplex* demodChanSiso;
hipFloatComplex* demodSig;
hipFloatComplex* demodSigFft;
hipfftHandle demodPlan;
hipFloatComplex* demodSigLlr;
int* demodDemapBpskL;
int* demodDemapQpskL;
int* demodDemap16QamL;
int* demodDemap64QamL;
int* demodDemapBpskNL;
int* demodDemapQpskNL;
int* demodDemap16QamNL;
int* demodDemap64QamNL;
int* demodDemap256QamNL;

__global__
void cuDemodChopSamp(int n, hipFloatComplex* sig, hipFloatComplex* sigfft)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = i / 80;       // symbol index
  int offset = i % 80;  
  if(i < n && offset >= 8 && offset < 72)
  {
    sigfft[j*64 + offset - 8] = sig[i];
  }
}

__global__
void cuDemodChanComp(int n, hipFloatComplex* sigfft, hipFloatComplex* chan)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int offset = i % 64;
  if(i < n)
  {
    sigfft[i] = hipCdivf(sigfft[i], chan[offset]);
  }
}

// __global__
// void cuDemodChanComp(int n, hipFloatComplex* sigfft, hipFloatComplex* chan)
// {
//   int i = threadIdx.x;
//   int I = blockIdx.x * blockDim.x + threadIdx.x;
//   int offset = I % 64;
//   __shared__ hipFloatComplex chanIn[64];
//   if(I >= n)
//   {
//     return;
//   }
//   if(i < 64)
//   {
//     chanIn[i] = chan[i];
//   }
//   __syncthreads();
//   sigfft[I] = hipCdivf(sigfft[I], chanIn[offset]);
// }

__global__
void cuDemodQamToLlrBpskL(int n, hipFloatComplex* sigfft, float* llr, int* demap)
{

}

void cuDemodMall()
{
  hipMalloc(&demodChanSiso, sizeof(hipFloatComplex) * 64);
  hipMalloc(&demodSig, sizeof(hipFloatComplex) * CUDEMOD_S_MAX * 80);
  hipMalloc(&demodSigFft, sizeof(hipFloatComplex) * CUDEMOD_S_MAX * 64);
  if(hipfftPlan1d(&demodPlan, 64, HIPFFT_C2C, CUDEMOD_FFT_BATCH) != HIPFFT_SUCCESS){
    std::cout<<"cloud80211 cufft, plan creation failed"<<std::endl;
  }
  hipMalloc(&demodSigLlr, sizeof(float) * CUDEMOD_S_MAX * 52 * 8);
  
  hipMalloc(&demodDemapBpskL, sizeof(int) * 48);
  hipMalloc(&demodDemapQpskL, sizeof(int) * 96);
  hipMalloc(&demodDemap16QamL, sizeof(int) * 192);
  hipMalloc(&demodDemap64QamL, sizeof(int) * 288);
  hipMalloc(&demodDemapBpskNL, sizeof(int) * 52);
  hipMalloc(&demodDemapQpskNL, sizeof(int) * 104);
  hipMalloc(&demodDemap16QamNL, sizeof(int) * 208);
  hipMalloc(&demodDemap64QamNL, sizeof(int) * 312);
  hipMalloc(&demodDemap256QamNL, sizeof(int) * 416);
  hipMemcpy(demodDemapBpskL, mapDeintLegacyBpsk, 48*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(demodDemapQpskL, mapDeintLegacyQpsk, 96*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(demodDemap16QamL, mapDeintLegacy16Qam, 192*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(demodDemap64QamL, mapDeintLegacy64Qam, 288*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(demodDemapBpskNL, mapDeintNonlegacyBpsk, 52*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(demodDemapQpskNL, mapDeintNonlegacyQpsk, 104*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(demodDemap16QamNL, mapDeintNonlegacy16Qam, 208*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(demodDemap64QamNL, mapDeintNonlegacy64Qam, 312*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(demodDemap256QamNL, mapDeintNonlegacy256Qam, 416*sizeof(int), hipMemcpyHostToDevice);
}

void cuDemodFree()
{
  hipFree(demodChanSiso);
  hipFree(demodSig);
  hipFree(demodSigFft);
  hipfftDestroy(demodPlan);
  hipFree(demodSigLlr);
  hipFree(demodDemapBpskL);
  hipFree(demodDemapQpskL);
  hipFree(demodDemap16QamL);
  hipFree(demodDemap64QamL);
  hipFree(demodDemapBpskNL);
  hipFree(demodDemapQpskNL);
  hipFree(demodDemap16QamNL);
  hipFree(demodDemap64QamNL);
  hipFree(demodDemap256QamNL);
}

void cuDemodChanSiso(hipFloatComplex *chan)
{
  hipMemcpy(demodChanSiso, chan, 64*sizeof(hipFloatComplex), hipMemcpyHostToDevice);
}

void cuDemodSigCopy(int i, int n, const hipFloatComplex *sig)
{
  if(i >= 0 && n >= 0 && (i+n) < (CUDEMOD_S_MAX * 80))
  {
    hipMemcpy(&demodSig[i], sig, n*sizeof(hipFloatComplex), hipMemcpyHostToDevice);
  }
}

void cuDemodSiso(c8p_mod* m)
{
  cuDemodChopSamp<<<(m->nSym * m->nSymSamp)/1024 + 1, 1024>>>(m->nSym * m->nSymSamp, demodSig, demodSigFft);
  for(int symIter=0; symIter < ((m->nSym + CUDEMOD_FFT_BATCH - 1) / CUDEMOD_FFT_BATCH); symIter++ )   // each round inlcudes 256 batches
  {
    hipfftExecC2C(demodPlan, &demodSigFft[symIter*CUDEMOD_FFT_BATCH*64], &demodSigFft[symIter*CUDEMOD_FFT_BATCH*64], HIPFFT_FORWARD);
  }
  cuDemodChanComp<<<(m->nSym * 64)/1024 + 1, 1024>>>(m->nSym * 64, demodSigFft, demodChanSiso);

}

void cuDemodDebug(int n, hipFloatComplex* outcomp, int m, float* outfloat)
{
  hipMemcpy(outcomp, demodSigFft, n*sizeof(hipFloatComplex), hipMemcpyDeviceToHost);
  // hipMemcpy(outfloat, demodSigFft, m*sizeof(float), hipMemcpyDeviceToHost);
}