#include "hip/hip_runtime.h"
/*
 *
 *     GNU Radio IEEE 802.11a/g/n/ac 20M bw and upto 2x2
 *     PHY utilization functions and parameters CUDA Version
 *     Copyright (C) Dec 1, 2022  Zelin Yun
 *
 *     This program is free software: you can redistribute it and/or modify
 *     it under the terms of the GNU Affero General Public License as published
 *     by the Free Software Foundation, either version 3 of the License, or
 *     (at your option) any later version.
 *
 *     This program is distributed in the hope that it will be useful,
 *     but WITHOUT ANY WARRANTY; without even the implied warranty of
 *     MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *     GNU Affero General Public License for more details.
 *
 *     You should have received a copy of the GNU Affero General Public License
 *     along with this program.  If not, see <https://www.gnu.org/licenses/>.
 */

#include "cloud80211phycu.cuh"

const int FFT_26_DEMAP[64] = {
	48, 24, 25, 26, 27, 28, 29, 49, 30, 31, 32, 33, 34, 35, 36, 37, 
	38, 39, 40, 41, 42, 50, 43, 44, 45, 46, 47, 51, 52, 53, 54, 55, 
	56, 57, 58, 59, 60, 61, 0, 1, 2, 3, 4, 62, 5, 6, 7, 8, 
	9, 10, 11, 12, 13, 14, 15, 16, 17, 63, 18, 19, 20, 21, 22, 23
};

const float LTF_L_26_F_FLOAT[64] = {
    0.0f, 1.0f, -1.0f, -1.0f, 
    1.0f, 1.0f, -1.0f, 1.0f, 
    -1.0f, 1.0f, -1.0f, -1.0f, 
    -1.0f, -1.0f, -1.0f, 1.0f, 
    1.0f, -1.0f, -1.0f, 1.0f, 
    -1.0f, 1.0f, -1.0f, 1.0f, 
    1.0f, 1.0f, 1.0f, 0.0f, 
    0.0f, 0.0f, 0.0f, 0.0f, 
    0.0f, 0.0f, 0.0f, 0.0f, 
    0.0f, 0.0f, 1.0f, 1.0f, 
    -1.0f, -1.0f, 1.0f, 1.0f, 
    -1.0f, 1.0f, -1.0f, 1.0f, 
    1.0f, 1.0f, 1.0f, 1.0f, 
    1.0f, -1.0f, -1.0f, 1.0f, 
    1.0f, -1.0f, 1.0f, -1.0f, 
    1.0f, 1.0f, 1.0f, 1.0f};

hipFloatComplex* signalX;
hipFloatComplex* signalY;
hipFloatComplex* signalA;
hipFloatComplex* signalB;
hipFloatComplex* signalHSig;
float* signalLLR;
hipfftHandle signalPlan;
hipFloatComplex* signalLtf;
int* signalLLRMap;

void signalMall()
{
  hipMalloc(&signalX, 8192*sizeof(hipFloatComplex));
  hipMalloc(&signalY, 8192*sizeof(hipFloatComplex));
  hipMalloc(&signalA, 240*sizeof(hipFloatComplex));
  hipMalloc(&signalB, 192*sizeof(hipFloatComplex));
  hipMalloc(&signalHSig, 128*sizeof(hipFloatComplex));
  hipMalloc(&signalLtf, 64*sizeof(hipFloatComplex));
  hipMalloc(&signalLLR, 64*sizeof(float));
  hipMalloc(&signalLLRMap, 64*sizeof(int));
  hipFloatComplex signalLtfTmp[64];
  for(int i=0;i<64;i++)
  {
    signalLtfTmp[i] = make_hipComplex(LTF_L_26_F_FLOAT[i]*2.0f, 0.0f);
  }
  hipMemcpy(signalLtf, signalLtfTmp, 64*sizeof(hipFloatComplex), hipMemcpyHostToDevice);
  hipMemcpy(signalLLRMap, FFT_26_DEMAP, 64*sizeof(int), hipMemcpyHostToDevice);
  hipfftPlan1d(&signalPlan, 64, HIPFFT_C2C, 3);
}

void signalFree()
{
  hipFree(signalX);
  hipFree(signalY);
  hipFree(signalA);
  hipFree(signalB);
  hipFree(signalHSig);
  hipFree(signalLtf);
  hipFree(signalLLR);
  hipFree(signalLLRMap);
  hipfftDestroy(signalPlan);
}

__global__
void cuSignalKernel(int n, int s, float radStep, hipFloatComplex *x, hipFloatComplex *y)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i < n)
  {
    y[i] = hipCmulf(x[i], make_hipFloatComplex(cosf( ((float)s+i) * radStep), sinf(((float)s+i) * radStep)));
  }
}

void cuSignalCfoCompen(int n, int s, float radStep, const hipFloatComplex *x, hipFloatComplex *y)
{
  hipMemcpy(signalX, x, n*sizeof(hipFloatComplex), hipMemcpyHostToDevice);
  // N+255 means requires at least 1 block, and 256 means 256 threads in each block
  cuSignalKernel<<<(n+1024)/1024, 1024>>>(n, s, radStep, signalX, signalY);
  hipMemcpy(y, signalY, n*sizeof(hipFloatComplex), hipMemcpyDeviceToHost);
}

__global__
void cuSignalChannelCfo(int s, float radStep, hipFloatComplex *x, hipFloatComplex *y)
{
  int i = threadIdx.x;
  if(i >= 8 && i < 72)
  {
    y[i-8] = hipCmulf(x[i], make_hipFloatComplex(cosf( ((float)s+i) * radStep), sinf(((float)s+i) * radStep)));
  }
  if(i >= 72 && i < 136)
  {
    y[i-8] = hipCmulf(x[i], make_hipFloatComplex(cosf( ((float)s+i) * radStep), sinf(((float)s+i) * radStep)));
  }
  if(i >= 152 && i < 216)
  {
    y[i-24] = hipCmulf(x[i], make_hipFloatComplex(cosf( ((float)s+i) * radStep), sinf(((float)s+i) * radStep)));
  }
}

__global__
void cuSignalChannel(hipFloatComplex* inSig, hipFloatComplex* hsig, hipFloatComplex* ltf, float*llr, int* demap)
{
  int i = threadIdx.x;
  hsig[i] = hipCdivf( hipCaddf(inSig[i], inSig[i+64]), ltf[i]);
  hsig[i + 64] = hipCdivf(inSig[i+128], hsig[i]);
  llr[demap[i]] = hipCrealf(hsig[i + 64]);
}

void cuSignalChannel(int s, float radStep, const hipFloatComplex *sig, hipFloatComplex *h, float* llr)
{
  hipMemcpy(signalA, sig, 240*sizeof(hipFloatComplex), hipMemcpyHostToDevice);
  cuSignalChannelCfo<<<1, 216>>>(s, radStep, signalA, signalB);
  hipfftExecC2C(signalPlan, signalB, signalB, HIPFFT_FORWARD);
  cuSignalChannel<<<1, 64>>>(signalB, signalHSig, signalLtf, signalLLR, signalLLRMap);
  hipMemcpy(h, signalHSig, 64*sizeof(hipFloatComplex), hipMemcpyDeviceToHost);
  hipMemcpy(llr, signalLLR, 48*sizeof(float), hipMemcpyDeviceToHost);
}



/*--------------------------------------------------------------------------------------------------------*/

hipFloatComplex* ppSig;
hipFloatComplex* ppSigConj;
hipFloatComplex* ppSigConjAvg;
float* ppSigConjAvgMag;
float* ppSigMag2;
float* ppSigMag2Avg;
float* ppOut;

hipFloatComplex ppTest[PREPROC_MAX];

void preprocMall()
{
  hipMalloc(&ppSig, PREPROC_MAX*sizeof(hipFloatComplex));
  hipMalloc(&ppSigConj, PREPROC_MAX*sizeof(hipFloatComplex));
  hipMalloc(&ppSigConjAvg, PREPROC_MAX*sizeof(hipFloatComplex));
  hipMalloc(&ppSigConjAvgMag, PREPROC_MAX*sizeof(float));
  hipMalloc(&ppSigMag2, PREPROC_MAX*sizeof(float));
  hipMalloc(&ppSigMag2Avg, PREPROC_MAX*sizeof(float));
  hipMalloc(&ppOut, PREPROC_MAX*sizeof(float));
}

void preprocFree()
{
  hipFree(ppSig);
  hipFree(ppSigConj);
  hipFree(ppSigConjAvg);
  hipFree(ppSigConjAvgMag);
  hipFree(ppSigMag2);
  hipFree(ppSigMag2Avg);
  hipFree(ppOut);
}

__global__
void cuPreProcConj(int n, hipFloatComplex* inSig, hipFloatComplex* inSigConj)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i < (n-16))
  {
    inSigConj[i] = hipCmulf(inSig[i], make_hipFloatComplex (hipCrealf(inSig[i+16]), -hipCimagf(inSig[i+16])));
  }
}

__global__
void cuPreProcMag2(int n, hipFloatComplex* inSig, float* inSigMag2)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i < n)
  {
    inSigMag2[i] = hipCabsf(inSig[i]);
    inSigMag2[i] = inSigMag2[i] * inSigMag2[i];
  }
}

__global__
void cuPreProcConjAvgMag(int n, hipFloatComplex* inSigConj, hipFloatComplex* inSigConjAvg, float* inSigConjAvgMag)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i < (n-48))
  {
    inSigConjAvg[i] = make_hipFloatComplex(0.0f, 0.0f);
    for(int j=0;j<48;j++)
    {
      inSigConjAvg[i] = hipCaddf(inSigConjAvg[i], inSigConj[i+j]);
    }
    inSigConjAvg[i] = hipCdivf(inSigConjAvg[i], make_hipFloatComplex(48.0f, 0.0f));
    inSigConjAvgMag[i] = hipCabsf(inSigConjAvg[i]);
  }
}

__global__
void cuPreProcMag2AvgOut(int n, float* inSigMag2, float* inSigMag2Avg, float* inSigConjAvgMag, float* out)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i < (n-80))
  {
    inSigMag2Avg[i+16] = 0.0f;
    for(int j=0;j<64;j++)
    {
      inSigMag2Avg[i+16] += inSigMag2[i+16+j];
    }
    inSigMag2Avg[i+16] = inSigMag2Avg[i+16] / 64.0f;
    out[i] = inSigConjAvgMag[i] / inSigMag2Avg[i+16];
  }
}

void cuPreProc(int n, const hipFloatComplex *sig, float* ac, hipFloatComplex* conj)
{
  if(n > 80)
  {
    // for(int i=0;i<n-16;i++)
    // {
    //   std::cout<<hipCrealf(sig[i])<<" ";
    // }
    hipMemcpy(ppSig, sig, n*sizeof(hipFloatComplex), hipMemcpyHostToDevice);
    cuPreProcConj<<<n/1024 + 1, 1024>>>(n, ppSig, ppSigConj);
    // hipMemcpy(ppTest, ppSigConj, n*sizeof(hipFloatComplex), hipMemcpyDeviceToHost);
    // for(int i=0;i<n-16;i++)
    // {
    //   std::cout<<hipCrealf(ppTest[i])<<" ";
    // }
    cuPreProcMag2<<<n/1024 + 1, 1024>>>(n, ppSig, ppSigMag2);
    cuPreProcConjAvgMag<<<n/1024 + 1, 1024>>>(n, ppSigConj, ppSigConjAvg, ppSigConjAvgMag);
    cuPreProcMag2AvgOut<<<n/1024 + 1, 1024>>>(n, ppSigMag2, ppSigMag2Avg, ppSigConjAvgMag, ppOut);
    hipMemcpy(ac, ppOut, (n - 80)*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(conj, ppSigConjAvg, (n - 80)*sizeof(hipFloatComplex), hipMemcpyDeviceToHost);
  }
}