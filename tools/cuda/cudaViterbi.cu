
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <chrono>
#include <iostream>

#define CUDEMOD_V_MAX 960 // max llr len

// viterbi, next state of each state with S1 = 0 and 1
const int SV_STATE_NEXT[64][2] = {
    { 0, 32 },  { 0, 32 },  { 1, 33 },  { 1, 33 },  { 2, 34 },  { 2, 34 },  { 3, 35 },
    { 3, 35 },  { 4, 36 },  { 4, 36 },  { 5, 37 },  { 5, 37 },  { 6, 38 },  { 6, 38 },
    { 7, 39 },  { 7, 39 },  { 8, 40 },  { 8, 40 },  { 9, 41 },  { 9, 41 },  { 10, 42 },
    { 10, 42 }, { 11, 43 }, { 11, 43 }, { 12, 44 }, { 12, 44 }, { 13, 45 }, { 13, 45 },
    { 14, 46 }, { 14, 46 }, { 15, 47 }, { 15, 47 }, { 16, 48 }, { 16, 48 }, { 17, 49 },
    { 17, 49 }, { 18, 50 }, { 18, 50 }, { 19, 51 }, { 19, 51 }, { 20, 52 }, { 20, 52 },
    { 21, 53 }, { 21, 53 }, { 22, 54 }, { 22, 54 }, { 23, 55 }, { 23, 55 }, { 24, 56 },
    { 24, 56 }, { 25, 57 }, { 25, 57 }, { 26, 58 }, { 26, 58 }, { 27, 59 }, { 27, 59 },
    { 28, 60 }, { 28, 60 }, { 29, 61 }, { 29, 61 }, { 30, 62 }, { 30, 62 }, { 31, 63 },
    { 31, 63 }
};

// viterbi, output coded 2 bits of each state with S1 = 0 and 1
const int SV_STATE_OUTPUT[64][2] = {
    { 0, 3 }, { 3, 0 }, { 2, 1 }, { 1, 2 }, { 0, 3 }, { 3, 0 }, { 2, 1 }, { 1, 2 },
    { 3, 0 }, { 0, 3 }, { 1, 2 }, { 2, 1 }, { 3, 0 }, { 0, 3 }, { 1, 2 }, { 2, 1 },
    { 3, 0 }, { 0, 3 }, { 1, 2 }, { 2, 1 }, { 3, 0 }, { 0, 3 }, { 1, 2 }, { 2, 1 },
    { 0, 3 }, { 3, 0 }, { 2, 1 }, { 1, 2 }, { 0, 3 }, { 3, 0 }, { 2, 1 }, { 1, 2 },
    { 1, 2 }, { 2, 1 }, { 3, 0 }, { 0, 3 }, { 1, 2 }, { 2, 1 }, { 3, 0 }, { 0, 3 },
    { 2, 1 }, { 1, 2 }, { 0, 3 }, { 3, 0 }, { 2, 1 }, { 1, 2 }, { 0, 3 }, { 3, 0 },
    { 2, 1 }, { 1, 2 }, { 0, 3 }, { 3, 0 }, { 2, 1 }, { 1, 2 }, { 0, 3 }, { 3, 0 },
    { 1, 2 }, { 2, 1 }, { 3, 0 }, { 0, 3 }, { 1, 2 }, { 2, 1 }, { 3, 0 }, { 0, 3 },
};

const int SV_PUNC_12[2] = { 1, 1 };
// const int SV_PUNC_23[4] = { 1, 1, 1, 0 };
// const int SV_PUNC_34[6] = { 1, 1, 1, 0, 0, 1 };
// const int SV_PUNC_56[10] = { 1, 1, 1, 0, 0, 1, 1, 0, 0, 1 };

__global__ void cuDecodeViterbi(float* llr,
                                int len,
                                int trellis,
                                int crlen,
                                int* punc,
                                int* s_his,
                                int* s_output,
                                int* s_next,
                                int* s_seq)
{
    int i = threadIdx.x;
    int v_cr_p = 0;
    int tmpUsed = 0;
    float *v_ae_pPre, *v_ae_pCur, *v_ae_pTmp;
    float v_acc_tmp0, v_acc_tmp1;
    int v_next0, v_next1;
    int v_t = 0;

    __shared__ float v_accum_err0[64];
    __shared__ float v_accum_err1[64];
    __shared__ float v_tab_t[4];

    if (i == 0) {
        v_accum_err0[i] = 0.0f;
    } else {
        v_accum_err0[i] = -1000000000000000.0f;
    }
    v_accum_err1[i] = -1000000000000000.0f;
    v_ae_pCur = v_accum_err1;
    v_ae_pPre = v_accum_err0;

    while ((tmpUsed + punc[v_cr_p] + punc[v_cr_p + 1]) <= len && v_t < trellis) {
        if (i == 0) {
            v_tab_t[0] = 0.0f;
            if (punc[v_cr_p]) {
                v_tab_t[2] = llr[tmpUsed];
                v_tab_t[3] = llr[tmpUsed];
                tmpUsed++;
            } else {
                v_tab_t[2] = 0.0f;
                v_tab_t[3] = 0.0f;
            }
            if (punc[v_cr_p + 1]) {
                v_tab_t[1] = llr[tmpUsed];
                v_tab_t[3] += llr[tmpUsed];
                tmpUsed++;
            } else {
                v_tab_t[1] = 0.0f;
            }
        }
        __syncthreads();

        v_acc_tmp0 = v_ae_pPre[i] + v_tab_t[s_output[i * 2]];
        v_acc_tmp1 = v_ae_pPre[i] + v_tab_t[s_output[i * 2 + 1]];

        if ((i % 2) == 0) {
            v_next0 = s_next[i * 2];
            v_next1 = s_next[i * 2 + 1];
            if (v_acc_tmp0 > v_ae_pCur[v_next0]) {
                v_ae_pCur[v_next0] = v_acc_tmp0;
                s_his[(v_t + 1) * 64 + v_next0] = i;
            }
            if (v_acc_tmp1 > v_ae_pCur[v_next1]) {
                v_ae_pCur[v_next1] = v_acc_tmp1;
                s_his[(v_t + 1) * 64 + v_next1] = i;
            }
        }
        __syncthreads();

        if ((i % 2) == 1) {
            v_next0 = s_next[i * 2];
            v_next1 = s_next[i * 2 + 1];
            if (v_acc_tmp0 > v_ae_pCur[v_next0]) {
                v_ae_pCur[v_next0] = v_acc_tmp0;
                s_his[(v_t + 1) * 64 + v_next0] = i;
            }
            if (v_acc_tmp1 > v_ae_pCur[v_next1]) {
                v_ae_pCur[v_next1] = v_acc_tmp1;
                s_his[(v_t + 1) * 64 + v_next1] = i;
            }
        }

        v_ae_pTmp = v_ae_pPre;
        v_ae_pPre = v_ae_pCur;
        v_ae_pCur = v_ae_pTmp;

        v_ae_pCur[i] = -1000000000000000.0f;

        v_cr_p += 2;
        if (v_cr_p >= crlen) {
            v_cr_p = 0;
        }
        
        v_t++;
    }

    // trace back
    if (i == 0) {
        s_seq[trellis] = 0;
        for (int j = trellis; j > 0; j--) {
            s_seq[j - 1] = s_his[j * 64 + s_seq[j]];
        }
    }
}

int main(void)
{
    float inputllr[48] = { 1.0928201751414013f,  1.0757566237512861f,  1.0443222344438439f,
                           -0.9381446904334033f, 1.0779025723635047f,  -0.9123096293726042f,
                           1.0938056178714068f,  1.0665362726332714f,  1.0363148583542974f,
                           -0.9515434541238799f, -0.9590006255492449f, 1.0962848724441732f,
                           -0.9026187027756123f, 1.0669992583689938f,  -0.9736616028811736f,
                           1.061441391797273f,   1.0778133212429195f,  1.0193412838821938f,
                           -0.957565789175884f,  1.0128408927100188f,  -0.9215097596323263f,
                           1.0986584025224821f,  1.0396421848955428f,  -0.9860899622285565f,
                           1.0254655330591547f,  -0.9734139829200796f, -0.9116625609934133f,
                           1.0143852928597692f,  -0.9545594221179006f, 1.0526777630409843f,
                           1.075059150655888f,   1.0532262849922545f,  -0.9290862730407128f,
                           -0.9518102443232885f, 1.0620442555472498f,  1.0270067102620282f,
                           -0.9252142704804616f, 1.0720527501137866f,  1.0654530942602567f,
                           1.071470291734725f,   1.076092476224666f,   1.0364141110673173f,
                           -0.9411547216481134f, -0.9634717335799121f, 1.0007634396548253f,
                           -0.9490875876327685f, 1.079757586068487f,   1.0648163041204708f };
    uint8_t uncodedBits[24] = {1, 1, 0, 1, 0, 0, 1, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0};
    int v_trellis = 24;
    int llrlen = 48;
    uint8_t v_decodedBits[CUDEMOD_V_MAX];
    // cpu params
    std::cout<<"cpu params"<<std::endl;
    float v_accum_err0[64];
    float v_accum_err1[64];
    float *v_ae_pPre, *v_ae_pCur;
    // int v_state_his[64][CUDEMOD_V_MAX + 1];
    int v_state_his[CUDEMOD_V_MAX + 1][64];
    int v_state_seq[CUDEMOD_V_MAX + 1];
    int v_op0, v_op1, v_next0, v_next1;
    float v_acc_tmp0, v_acc_tmp1, v_t0, v_t1;
    float v_tab_t[4];
    const int* v_cr_punc;
    int v_cr_p, v_cr_len;
    int v_t;
    // cpu init
    std::cout<<"cpu init"<<std::endl;
    for (int i = 0; i < 64; i++) {
        for (int j = 0; j <= v_trellis; j++) {
            v_state_his[j][i] = 0;
        }
        v_accum_err0[i] = -1000000000000000.0f;
        v_accum_err1[i] = -1000000000000000.0f;
    }
    v_accum_err0[0] = 0;
    v_ae_pCur = &v_accum_err1[0];
    v_ae_pPre = &v_accum_err0[0];
    v_t = 0;
    v_cr_p = 0;
    v_cr_len = 2;
    v_cr_punc = SV_PUNC_12;
    // cpu decode
    int tmpUsed = 0;
    while ((tmpUsed + v_cr_punc[v_cr_p] + v_cr_punc[v_cr_p + 1]) <= llrlen) {
        std::cout<<"cpu decode v_t: "<<v_t<<std::endl;
        if (v_cr_punc[v_cr_p]) {
            v_t0 = inputllr[tmpUsed];
            tmpUsed++;
        } else {
            v_t0 = 0.0f;
        }
        if (v_cr_punc[v_cr_p + 1]) {
            v_t1 = inputllr[tmpUsed];
            tmpUsed++;
        } else {
            v_t1 = 0.0f;
        }

        v_tab_t[0] = 0.0f;
        v_tab_t[1] = v_t1;
        v_tab_t[2] = v_t0;
        v_tab_t[3] = v_t1 + v_t0;

        /* repeat for each possible state */
        for (int i = 0; i < 64; i++) {
            v_op0 = SV_STATE_OUTPUT[i][0];
            v_op1 = SV_STATE_OUTPUT[i][1];

            v_acc_tmp0 = v_ae_pPre[i] + v_tab_t[v_op0];
            v_acc_tmp1 = v_ae_pPre[i] + v_tab_t[v_op1];

            v_next0 = SV_STATE_NEXT[i][0];
            v_next1 = SV_STATE_NEXT[i][1];

            if (v_acc_tmp0 > v_ae_pCur[v_next0]) {
                v_ae_pCur[v_next0] = v_acc_tmp0;
                v_state_his[v_t + 1][v_next0] = i;
            }

            if (v_acc_tmp1 > v_ae_pCur[v_next1]) {
                v_ae_pCur[v_next1] = v_acc_tmp1;
                v_state_his[v_t + 1][v_next1] = i;
            }
        }

        /* update accum_err_metric array */
        float* tmp = v_ae_pPre;
        v_ae_pPre = v_ae_pCur;
        v_ae_pCur = tmp;

        for (int i = 0; i < 64; i++) {
            v_ae_pCur[i] = -1000000000000000.0f;
        }
        v_cr_p += 2;
        if (v_cr_p >= v_cr_len) {
            v_cr_p = 0;
        }

        v_t++;
        if (v_t >= v_trellis) {
            break;
        }
    }
    // std::cout<<"cpu history"<<std::endl;
    // for(int i=0;i<25;i++)
    // {
    //     for(int j=0;j<64;j++)
    //         std::cout<<v_state_his[i][j]<<", ";
    //     std::cout<<std::endl;
    // }
    // std::cout<<std::endl;

    // std::cout<<"cpu state next"<<std::endl;
    // for(int i=0;i<64;i++)
    // {
    //     std::cout<<SV_STATE_NEXT[i][0]<<", ";
    //     std::cout<<SV_STATE_NEXT[i][1]<<", ";
    //     std::cout<<std::endl;
    // }
    // std::cout<<std::endl;

    std::cout<<"cpu trace back, state seq"<<std::endl;
    v_state_seq[v_trellis] = 0;
    for (int j = v_trellis; j > 0; j--) {
        v_state_seq[j - 1] = v_state_his[j][v_state_seq[j]];
    }
    for(int i=0;i<=v_trellis;i++)
    {
        std::cout<<v_state_seq[i]<<", ";
    }
    std::cout<<std::endl;

    for (int j = 0; j < v_trellis; j++) {
        if (v_state_seq[j + 1] == SV_STATE_NEXT[v_state_seq[j]][1]) {
            v_decodedBits[j] = 1;
        } else {
            v_decodedBits[j] = 0;
        }
    }

    int totalErrorNum = 0;
    std::cout<<"cpu decoded bits"<<std::endl;
    for(int i=0;i<24;i++)
    {
        std::cout<<(int)v_decodedBits[i]<<", ";
    }
    std::cout<<std::endl;
    std::cout<<"cpu uncoded bits"<<std::endl;
    for(int i=0;i<24;i++)
    {
        std::cout<<(int)uncodedBits[i]<<", ";
        if(v_decodedBits[i] != uncodedBits[i])
        {
            totalErrorNum++;
        }
    }
    std::cout<<std::endl;
    std::cout<<"cpu decoded error bits num: "<<totalErrorNum<<std::endl;


    float* cuv_llr;
    int* cuv_seq;
    int* cuv_bits;
    int* cuv_state_his;
    int* cuv_state_bit;
    int* cuv_state_next;
    int* cuv_state_output;
    int cpu_state_output[128];
    int* cuv_cr_punc;
    hipMalloc(&cuv_llr, sizeof(float) * CUDEMOD_V_MAX * 2);
    hipMemset(cuv_llr, 0, sizeof(float) * CUDEMOD_V_MAX * 2);
    hipMalloc(&cuv_seq, sizeof(int) * CUDEMOD_V_MAX);
    hipMalloc(&cuv_bits, sizeof(int) * CUDEMOD_V_MAX);
    hipMalloc(&cuv_state_his, sizeof(int) * 64 * (CUDEMOD_V_MAX + 1));
    hipMalloc(&cuv_state_bit, sizeof(int) * (CUDEMOD_V_MAX + 1));
    hipMalloc(&cuv_state_next, sizeof(int) * 128);
    hipMemcpy(cuv_state_next, SV_STATE_NEXT, 128 * sizeof(int), hipMemcpyHostToDevice);
    hipMalloc(&cuv_state_output, sizeof(int) * 128);
    hipMemcpy(cuv_state_output, SV_STATE_OUTPUT, 128 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(cpu_state_output, cuv_state_next, 128 * sizeof(int), hipMemcpyDeviceToHost);
    hipMalloc(&cuv_cr_punc, sizeof(int) * 22);
    int tmpPunc[22] = {
        1, 1, 1, 1, 1, 0, 1, 1, 1, 0, 0, 1, 1, 1, 1, 0, 0, 1, 1, 0, 0, 1
    };
    hipMemcpy(cuv_cr_punc, tmpPunc, 22 * sizeof(int), hipMemcpyHostToDevice);

    hipMemcpy(cuv_llr, inputllr, 48 * sizeof(float), hipMemcpyHostToDevice);
    cuDecodeViterbi<<<1,64>>>(cuv_llr, 48, v_trellis, 2, cuv_cr_punc, cuv_state_his, cuv_state_output, cuv_state_next, cuv_seq);

    float cpu_float[CUDEMOD_V_MAX * 2];
    int cpu_int[CUDEMOD_V_MAX * 2];
    int cpu_his[CUDEMOD_V_MAX + 1][64];
    hipMemcpy(cpu_float, cuv_llr, sizeof(float) * CUDEMOD_V_MAX * 2, hipMemcpyDeviceToHost);
    hipMemcpy(cpu_int, cuv_seq, sizeof(int) * CUDEMOD_V_MAX, hipMemcpyDeviceToHost);
    hipMemcpy(cpu_his, cuv_state_his, sizeof(int) * 64 * (CUDEMOD_V_MAX + 1), hipMemcpyDeviceToHost);
    // std::cout<<"cuda debug float"<<std::endl;
    // for(int i=0;i<100;i++)
    // {
    //     std::cout<<i<<" "<<cpu_float[i]<<std::endl;
    // }

    std::cout<<"cuda debug int"<<std::endl;
    for(int i=0;i<48;i++)
    {
        std::cout<<cpu_int[i]<<", ";
    }
    std::cout<<std::endl;

    // std::cout<<"cuda history"<<std::endl;
    // for(int i=0;i<25;i++)
    // {
    //     for(int j=0;j<64;j++)
    //         std::cout<<cpu_his[i][j]<<", ";
    //     std::cout<<std::endl;
    // }
    // std::cout<<std::endl;

    // std::cout<<"cuda state next"<<std::endl;
    // for(int i=0;i<64;i++)
    // {
    //     std::cout<<cpu_state_output[i*2]<<", ";
    //     std::cout<<cpu_state_output[i*2+1]<<", ";
    //     std::cout<<std::endl;
    // }
    // std::cout<<std::endl;

    hipFree(cuv_llr);
    hipFree(cuv_seq);
    hipFree(cuv_bits);
    hipFree(cuv_state_his);
    hipFree(cuv_state_bit);
    hipFree(cuv_state_next);
    hipFree(cuv_state_output);
    hipFree(cuv_cr_punc);
    return 0;
}